#include "hip/hip_runtime.h"
//
// Created by acacia on 10/23/21.
//

#include "mpm3d.cuh"

using namespace utils;

namespace mpm
{
    using Real = float;
    using Vector = Eigen::Vector2f;
    using Matrix = Eigen::Matrix2f;
    using Vectori = Eigen::Vector2i;

    constexpr __device__ int dim = 2, n_grid = 128, steps = 20;
    constexpr __device__ Real dt = 2e-4;
    constexpr __device__ int n_particles =
            power(n_grid, dim) / power(2, dim - 1);
    constexpr __device__ int neighbour = power(3, dim);

    constexpr __device__ Real dx = 1.0 / n_grid;
    constexpr Real p_rho = 1.0;
    constexpr Real p_vol = power(dx * 0.5, 2);
    constexpr __device__ Real p_mass = p_vol * p_rho;
    constexpr __device__ Real gravity = 9.8;
    constexpr __device__ int bound = 3;
    constexpr __device__ Real E = 400;

    Vector* x_dev;
    Vector* v_dev;
    Matrix* C_dev;
    Real* J_dev;
    Vector* grid_v_dev;
    Real* grid_m_dev;

    int max_threads_per_block;

    template<class R, class A>
    __device__ R narrow_cast(const A& a)
    {
        R r = R(a);
        if (A(r) != a) printf("warning: info loss in narrow_cast");
        return r;
    }

    __device__ Vectori get_indices(size_t idx)
    {
        Vectori indices;
        for (auto i = dim - 1; i >= 0; i--)
        {
            indices[i] = narrow_cast<int, size_t>(idx % 3 - 1);
            idx /= 3;
        }
        return indices;
    }

    __global__ void init_kernel(Real* J)
    {
        auto idx = blockIdx.x * blockDim.x + threadIdx.x;
        J[idx] = 1;
    }

    __global__ void reset_kernel(Vector* grid_v, Real* grid_m)
    {
        auto idx = blockIdx.x * blockDim.x + threadIdx.x;
        grid_v[idx].setZero();
        grid_m[idx] = 0;
    }

    __global__ void
    particle_to_grid_kernel(Vector* x, Vector* v, Matrix* C, const Real* J,
            Vector* grid_v, Real* grid_m)
    {
        auto idx = blockIdx.x * blockDim.x + threadIdx.x;
        // do not use the auto keyword with Eigen's expressions
        Vector Xp = x[idx] / dx;
        Vectori base = (Xp.array() - 0.5).cast<int>();
        Vector fx = Xp - base.cast<Real>();
        std::array<Vector, 3> w{ 0.5 * (1.5 - fx.array()).pow(2),
                                 0.75 - (fx.array() - 1.0).pow(2),
                                 0.5 * (fx.array() - 0.5).pow(2) };
        auto stress = -dt * 4 * E * p_vol * (J[idx] - 1) / pow(dx, 2);
        Matrix affine = Matrix::Identity() * stress + p_mass * C[idx];
        for (auto offset_idx = 0; offset_idx < neighbour; offset_idx++)
        {
            Vectori offset = get_indices(offset_idx).array() - 1;
            Vector dpos = (offset.cast<Real>() - fx) * dx;
            Real weight = 1.0;
            for (auto i = 0; i < dim; i++)
            {
                weight *= w[offset[i]][i];
            }

            // TODO: evaluate performance of atomic operations
            Vector grid_v_add = weight * (p_mass * v[idx] + affine * dpos);
            auto grid_m_add = weight * p_mass;
            Vectori grid_idx_vector = base + offset;
            auto grid_idx = 0;
            for (auto i = 0; i < dim; i++)
            {
                grid_idx = grid_idx * n_grid + grid_idx_vector[i];
            }
            for (auto i = 0; i < dim; i++)
            {
                atomicAdd(&(grid_v[grid_idx][i]), grid_v_add[i]);
            }
            atomicAdd(&(grid_m[grid_idx]), grid_m_add);
        }
    }

    __global__ void grid_update_kernel(Vector* grid_v, Real* grid_m)
    {
        auto idx = blockDim.x * blockIdx.x + threadIdx.x;
        if (grid_m[idx] > 0)
        {
            grid_v[idx] /= grid_m[idx];
        }
        grid_v[idx][1] -= dt * gravity;
        Vectori indices = get_indices(idx);
        for (auto i = 0; i < dim; i++)
        {
            if ((indices[i] < bound && grid_v[idx][i] < 0) ||
                (indices[i] > n_grid - bound && grid_v[idx][i] > 0))
            {
                grid_v[idx][i] = 0;
            }
        }
    }

    __global__ void
    grid_to_particle_kernel(Vector* x, Vector* v, Matrix* C, Real* J,
            Vector* grid_v)
    {
        auto idx = blockIdx.x * blockDim.x + threadIdx.x;
        Vector Xp = x[idx] / dx;
        Vectori base = (Xp.array() - 0.5).cast<int>();
        Vector fx = Xp - base.cast<Real>();
        std::array<Vector, 3> w{ 0.5 * (1.5 - fx.array()).pow(2),
                                 0.75 - (fx.array() - 1.0).pow(2),
                                 0.5 * (fx.array() - 0.5).pow(2) };

        Vector new_v = Vector::Zero();
        Matrix new_C = Matrix::Zero();
        for (auto offset_idx = 0; offset_idx < neighbour; offset_idx++)
        {
            Vectori offset = get_indices(offset_idx).array() - 1;
            Vector dpos = (offset.cast<Real>() - fx) * dx;
            Real weight = 1.0;
            for (auto i = 0; i < dim; i++)
            {
                weight *= w[offset[i]][i];
            }
            Vectori grid_idx_vector = base + offset;
            auto grid_idx = 0;
            for (auto i = 0; i < dim; i++)
            {
                grid_idx = grid_idx * n_grid + grid_idx_vector[i];
            }
            new_v += weight * grid_v[grid_idx];
            new_C += 4.0 * weight * grid_v[grid_idx] * dpos.transpose() /
                     pow(dx, 2);
        }
        v[idx] = new_v;
        x[idx] += dt * v[idx];
        J[idx] *= Real(1.0) + dt * new_C.trace();
        C[idx] = new_C;
    }

    __host__ void init()
    {
        hipFree(x_dev);
        hipFree(v_dev);
        hipFree(C_dev);
        hipFree(J_dev);
        hipFree(grid_v_dev);
        hipFree(grid_m_dev);

        hipMalloc(&x_dev, n_particles * sizeof(Vector));
        hipMalloc(&v_dev, n_particles * sizeof(Vector));
        hipMalloc(&C_dev, n_particles * sizeof(Matrix));
        hipMalloc(&J_dev, n_particles * sizeof(Real));
        hipMalloc(&grid_v_dev, power(n_grid, dim) * sizeof(Vector));
        hipMalloc(&grid_m_dev, power(n_grid, dim) * sizeof(Real));
        cuda_check_error();

        // initialize x on host and copy to device
        auto x_host = std::make_unique<Vector[]>(n_particles);
        for (auto i = 0; i < n_particles; i++)
        {
            x_host[i].setRandom();
            x_host[i] = (x_host[i] * 0.5).array() + 1.0;
            x_host[i] = (x_host[i] * 0.4).array() + 0.15;
        }
        hipMemcpy(x_dev, x_host.get(), n_particles * sizeof(Vector),
                hipMemcpyHostToDevice);

        hipDeviceProp_t prop{};
        hipGetDeviceProperties(&prop, 0);
        max_threads_per_block = prop.maxThreadsPerBlock;
        int block_num = get_block_num(n_particles,
                max_threads_per_block);
        init_kernel<<<block_num, max_threads_per_block>>>(J_dev);
        cuda_check_error();
    }

    void advance()
    {
        auto T = steps;
        int particle_block_num = get_block_num(n_particles,
                max_threads_per_block);
        int grid_block_num = get_block_num(power(n_grid, dim),
                max_threads_per_block);
        while (T--)
        {
            reset_kernel<<<grid_block_num, max_threads_per_block>>>(grid_v_dev,
                    grid_m_dev);

            particle_to_grid_kernel<<<particle_block_num, max_threads_per_block>>>(
                    x_dev, v_dev, C_dev, J_dev, grid_v_dev, grid_m_dev);

            grid_update_kernel<<<grid_block_num, max_threads_per_block>>>(
                    grid_v_dev, grid_m_dev);

            grid_to_particle_kernel<<<particle_block_num, max_threads_per_block>>>(
                    x_dev, v_dev, C_dev, J_dev, grid_v_dev);

            cuda_check_error();
        }
    }

    std::unique_ptr<Vector[]> to_numpy()
    {
        auto x_host = std::make_unique<Vector[]>(n_particles);
        hipMemcpy(x_host.get(), x_dev, n_particles * sizeof(Vector),
                hipMemcpyDeviceToHost);
        return x_host;
    }
}